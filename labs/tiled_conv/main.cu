#include "hip/hip_runtime.h"
#include "helper.hpp"


// Sequential code for the forward path of the convolution layer
// You should not modify this code
static void conv_forward_valid(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
                               const shape &ydims) {
  std::fill(Y, Y + ydims.flattened_length(), 0);

  for (auto i : range(0, ydims.num)) {
    for (auto m : range(0, ydims.depth )) {   // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width )) {
          const auto yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth )) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {   // filter height
              for (auto q : range(0, wdims.width )) { // filter width
                const auto xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const auto woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Baseline GPU kernel code for forward convolution.
// One thread per output index
// You should not modify this kernel as it is used for correctness comparison.
// Instead, define a new one below
__global__ void conv_forward_baseline_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
                                    const shape ydims) {


  const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = gx; i < ydims.num * ydims.depth * ydims.height * ydims.width; i += blockDim.x * gridDim.x) {
    Y[i] = 0.f;
  }

  for (size_t i = gx; i < ydims.num; i += gridDim.x * blockDim.x) {
    for (auto m : range(0, ydims.depth )) { // for each output feature map
      for (auto h : range(0, ydims.height)) { // for each output element
        for (auto w : range(0, ydims.width )) {
          const size_t yoffset = ((i * ydims.depth + m) * ydims.height + h) * ydims.width + w;
          for (auto c : range(0, xdims.depth )) {     // sum over all input feature maps
            for (auto p : range(0, wdims.height)) {   // filter height
              for (auto q : range(0, wdims.width )) { // filter width
                const size_t xoffset = ((((i * xdims.depth) + c) * xdims.height) + (h + p)) * xdims.width + (w + q);
                const size_t woffset = ((((m * wdims.depth) + c) * wdims.height) + p) * wdims.width + q;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_baseline(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
  const shape &ydims) {

  dim3 dimGrid(1);
  dim3 dimBlock(32);

  conv_forward_baseline_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());

}

// Implement your optimized kernel here.
// Make any modifications you wish.
// Don't forget to modify the host code below, if needed!
__global__ void conv_forward_opt_kernel(const float *X, const shape xdims, const float *W, const shape wdims, float *Y,
  const shape ydims) {

    const size_t gx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = gx; i < ydims.num * ydims.depth * ydims.height * ydims.width; i += blockDim.x * gridDim.x) {
    Y[i] = 0.f;
  }

  //@@ YOUR CODE HERE!
}

// Host code to configure baseline GPU kernel
static void convlayer_gpu_opt(const float *X, const shape &xdims, const float *W, const shape &wdims, float *Y,
  const shape &ydims) {

  // Modify this code to configure your optimized kernel.
  //@@ YOUR CODE HERE!!!
  dim3 dimGrid(1);
  dim3 dimBlock(32);
  conv_forward_opt_kernel<<<dimGrid, dimBlock>>>(X, xdims, W, wdims, Y, ydims);
  THROW_IF_ERROR(hipGetLastError());

}


static int eval(const shape wDims, const shape xDims) {

  // Generate model
  const auto conf_info = std::string("conv[wDims:") + std::to_string(wDims.num) + "," +
                                                      std::to_string(wDims.depth) + "," +
                                                      std::to_string(wDims.height) + "," +
                                                      std::to_string(wDims.width) +
                                                      " xDims:" + std::to_string(xDims.num) + "," +
                                                      std::to_string(xDims.depth) + "," +
                                                      std::to_string(xDims.height) + "," +
                                                      std::to_string(xDims.width) + "]";
  INFO("Running "  << conf_info);

  // Generate convolution weights
  float *hostW = allocate<float>(wDims);
  generate_convfilters(hostW, wDims);

  // generate input feature map
  float *hostX = allocate<float>(xDims);
  generate_data(hostX, xDims);

  // generate output feature map for verification
  const shape ydims = {xDims.num, wDims.num, (xDims.height - wDims.height + 1),
      (xDims.width - wDims.width + 1)};
  INFO("Allocating output tensor [" << ydims.num << "," << ydims.depth << "," << ydims.height << "," << ydims.width << "]");
  float *hostY = allocate<float>(ydims);
  float *expected = allocate<float>(ydims);
  generate_data(hostY, ydims);


  const size_t wByteCount = wDims.flattened_length() * sizeof(float);
  const size_t xByteCount = xDims.flattened_length() * sizeof(float);
  const size_t yByteCount = ydims.flattened_length() * sizeof(float);

  float *deviceW = nullptr, *deviceX = nullptr, *deviceY = nullptr;
  timer_start("Allocating GPU memory.");
  THROW_IF_ERROR(hipMalloc((void **)&deviceW, wByteCount));
  THROW_IF_ERROR(hipMalloc((void **)&deviceX, xByteCount));
  THROW_IF_ERROR(hipMalloc((void **)&deviceY, yByteCount));
  timer_stop();


  timer_start("Copying inputs to the GPU.");
  THROW_IF_ERROR(hipMemcpy(deviceW, hostW, wByteCount, hipMemcpyDefault));
  THROW_IF_ERROR(hipMemcpy(deviceX, hostX, xByteCount, hipMemcpyDefault));
  timer_stop();

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  timer_start("Performing GPU convlayer");
  convlayer_gpu_opt(deviceX, xDims, deviceW, wDims, deviceY, ydims);
  THROW_IF_ERROR(hipDeviceSynchronize());
  timer_stop();

  timer_start("Copying output to the CPU");
  THROW_IF_ERROR(hipMemcpy(hostY, deviceY, yByteCount, hipMemcpyDefault));
  timer_stop();

  // verify with provided implementation
  convlayer_gpu_baseline(deviceX, xDims, deviceW, wDims, deviceY, ydims);
  THROW_IF_ERROR(hipDeviceSynchronize());
  THROW_IF_ERROR(hipMemcpy(expected, deviceY, yByteCount, hipMemcpyDefault));
  // conv_forward_valid(hostX, xDims, hostW, wDims, expected, ydims);
  verify(expected, hostY, ydims);

  THROW_IF_ERROR(hipFree(deviceW));
  THROW_IF_ERROR(hipFree(deviceX));
  THROW_IF_ERROR(hipFree(deviceY));
  free(hostW);
  free(hostX);
  free(hostY);
  free(expected);

  return 0;
}



TEST_CASE("Convlayer", "[convlayer]") {
  SECTION("[wDims:0,0,0,0 xDims:100,1,32,32]") {
    eval({0,0,0,0}, {100,1,32,32});
  }
  SECTION("[wDims:1,1,1,1 xDims:100,1,32,32]") {
    eval({1,1,1,1}, {100,1,32,32});
  }
  SECTION("[wDims:32,1,5,5 xDims:1000,1,28,28]") {
    eval({32,1,5,5}, {1000,1,28,28});
  }
  SECTION("[wDims:16,1,3,3 xDims:100,1,32,32]") {
    eval({16,1,3,3}, {100,1,32,32});
  }

}
