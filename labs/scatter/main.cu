#include "hip/hip_runtime.h"
#include "helper.hpp"

__global__ void s2g_gpu_scatter_kernel(uint32_t *in, uint32_t *out, int len) {
  //@@ INSERT KERNEL CODE HERE
}

static void s2g_cpu_scatter(uint32_t *in, uint32_t *out, int len) {

  for (int inIdx = 0; inIdx < len; ++inIdx) {
    uint32_t intermediate = outInvariant(in[inIdx]);
    for (int outIdx = 0; outIdx < len; ++outIdx) {
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}

static void s2g_gpu_scatter(uint32_t *in, uint32_t *out, int len) {
  //@@ INSERT CODE HERE
}

static int eval(int inputLength) {
  uint32_t *deviceInput = nullptr;
  uint32_t *deviceOutput= nullptr;

  const std::string conf_info =
      std::string("scatter[len:") + std::to_string(inputLength) + "]";
  INFO("Running "  << conf_info);

  auto hostInput = generate_input(inputLength);

  const size_t byteCount = inputLength * sizeof(uint32_t);

  timer_start("Allocating GPU memory.");
  THROW_IF_ERROR(hipMalloc((void **)&deviceInput, byteCount));
  THROW_IF_ERROR(hipMalloc((void **)&deviceOutput, byteCount));
  timer_stop();

  timer_start("Copying input memory to the GPU.");
  THROW_IF_ERROR(hipMemcpy(deviceInput, hostInput.data(), byteCount,
                     hipMemcpyHostToDevice));
  THROW_IF_ERROR(hipMemset(deviceOutput, 0, byteCount));
  timer_stop();


  //////////////////////////////////////////
  // GPU Scatter Computation
  //////////////////////////////////////////
  timer_start( "Performing GPU Scatter computation");
  s2g_gpu_scatter(deviceInput, deviceOutput, inputLength);
  timer_stop();

  std::vector<uint32_t> hostOutput(inputLength);

  timer_start( "Copying output memory to the CPU");
  THROW_IF_ERROR(hipMemcpy(hostOutput.data(), deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  timer_stop();

  auto expected = compute_output(hostInput, inputLength);
  verify(expected, hostOutput);

  hipFree(deviceInput);
  hipFree(deviceOutput);

  return 0;
}

TEST_CASE("Scatter", "[scatter]") {
  SECTION("[inputSize:1024]") {
    eval(1024);
  }
  SECTION("[inputSize:2048]") {
    eval(2048);
  }
  SECTION("[inputSize:2047]") {
    eval(2047);
  }
  SECTION("[inputSize:2049]") {
    eval(2049);
  }
  SECTION("[inputSize:9101]") {
    eval(9101);
  }
  SECTION("[inputSize:9910]") {
    eval(9910);
  }
  SECTION("[inputSize:8192]") {
    eval(8192);
  }
  SECTION("[inputSize:8193]") {
    eval(8193);
  }
  SECTION("[inputSize:8191]") {
    eval(8191);
  }
  SECTION("[inputSize:16191]") {
    eval(16191);
  }
}
